#include "hip/hip_runtime.h"
﻿
#include "hip/hip_runtime.h"
#include ""

#include <stdio.h>
#include <sstream>
#include <fstream>
#include <vector>
#include <picojson.h>
//#include <iomanip>

struct Student
{
    double grade = 0.0;
    char name[20];
    int year = 0;
};

std::vector<Student> jsonToStudentVector(picojson::value jf) {
    picojson::array jsonarray = jf.get<picojson::array>();
    int size = jsonarray.size();
    std::vector<Student> students(size);
    for (int i = 0; i < size; i++)
    {
        Student stud = Student();
        stud.grade = jsonarray[i].get("grade").get<double>();
        std::string nameStr = jsonarray[i].get("name").get<std::string>();
        strcpy(stud.name, nameStr.c_str());
        stud.year = (int)jsonarray[i].get("year").get<double>();
        students[i] = stud;
    }
    return students;
}

__device__ char* studentToString(Student student) {
    int from = 0; 
    int to = 0;
    while (student.name[to] != '\0') to++;

    char* string = new char[to+10];
    string[from++] = '{'; to++;
    for (int i = from; i < to; i++) {
        string[i] = student.name[i - 1];
    }
    string[to++] = ' ';
    string[to++] = student.year + 48;
    string[to++] = ' ';

    int grade = student.grade * 100;
    for (int i = 3; i > 0; i--) {
        if(i == 1) string[to++] = '.';
        int div = (int)pow(10.0, i);
        int highest = floor((float)grade / div);
        string[to++] = highest + 48;
        grade -= highest*div;
    }
    string[to++] = grade + 48;

    string[to++] = '}';
    string[to] = '\0';

    return string;
}

__device__ void hashStudent(char* student, char* hash)
{
    int count = 0;
    while (student[count] != '\0') count++;
    int number = 0;
    for (int i = 0; i < count; i++) {
        number += i * student[i];
    }

    for (int i = 0; i < 40; i++) {
        int encoder = number % 36;
        if (encoder < 10) {
            hash[i] = encoder + 48;
        }
        else {
            hash[i] = encoder - 10 + 65;
        }
        number -= number % (count + encoder) - i;
    }
    hash[40] = '\0';
}

__device__ int countHashDigits(char* hash) {
    int count = 0;
    for (int i = 0; i < 40; i++)
        if (hash[i] > 47 && hash[i] < 58)
            count++;
    return count;
}

__global__ void hashCounts(Student* data, const int* count, char* results, int* index, char* hashes, int* hashIndex);

int main()
{
    std::vector<std::string> dataFiles = {
        "IFF-8-11_MikalauskasE_L1_dat_1.json",
        "IFF-8-11_MikalauskasE_L1_dat_2.json",
        "IFF-8-11_MikalauskasE_L1_dat_3.json",
    };

    std::vector<std::string> resFiles = {
        "IFF-8-11_MikalauskasE_L1_rez_1.txt",
        "IFF-8-11_MikalauskasE_L1_rez_2.txt",
        "IFF-8-11_MikalauskasE_L1_rez_3.txt",
    };

    hipDeviceProp_t prop{};
    hipGetDeviceProperties(&prop, 0);

    for (int i = 0; i < (int)dataFiles.size(); i++) {
        std::ifstream ifs(dataFiles[i]);
        picojson::value jf;
        picojson::parse(jf, ifs);
        std::vector<Student> studentVector = jsonToStudentVector(jf);
        int count = studentVector.size();
        Student* students = new Student[count]; 
        copy(studentVector.begin(), studentVector.end(), students);
        char* results = new char[count * 23+1];
        int index = 0;
        char* hashes = new char[count * 61 + 1];
        int hashIndex = 0;

        Student* device_students;
        int* device_count;
        char* device_results;
        int* device_index = 0;
        int size = count * sizeof(Student);
        char* device_hashes;
        int* device_hashIndex = 0;
        hipMalloc(&device_students, size);
        hipMalloc(&device_count, sizeof(int));
        hipMalloc(&device_results, sizeof(char)*count * 23 + 1);
        hipMalloc(&device_index, sizeof(int));
        hipMalloc(&device_hashes, sizeof(char) * count * 61 + 1);
        hipMalloc(&device_hashIndex, sizeof(int));

        hipMemcpy(device_students, students, size, hipMemcpyHostToDevice);
        hipMemcpy(device_count, &count, sizeof(int), hipMemcpyHostToDevice);
        hipMemcpy(device_results, results, sizeof(char) * count*23 +1, hipMemcpyHostToDevice);
        hipMemcpy(device_index, &index, sizeof(int), hipMemcpyHostToDevice);
        // hashes for printing
        hipMemcpy(device_hashes, hashes, sizeof(char) * count * 61 + 1, hipMemcpyHostToDevice);
        hipMemcpy(device_hashIndex, &index, sizeof(int), hipMemcpyHostToDevice);

        hashCounts <<<1, count/3 >>>(device_students, device_count, device_results, device_index, device_hashes, device_hashIndex);
        hipDeviceSynchronize();

        hipMemcpy(results, device_results, sizeof(char) * count * 23+1, hipMemcpyDeviceToHost);
        hipMemcpy(&index, device_index, sizeof(int), hipMemcpyDeviceToHost);
        hipMemcpy(hashes, device_hashes, sizeof(char) * count * 61 + 1, hipMemcpyDeviceToHost);
        hipMemcpy(&hashIndex, device_hashIndex, sizeof(int), hipMemcpyDeviceToHost);
        hipFree(device_students);
        hipFree(device_count);
        hipFree(device_results);
        hipFree(device_index);
        hipFree(device_hashes);
        hipFree(device_hashIndex);
        results[index] = '\0';
        hashes[hashIndex] = '\0';


        std::ofstream ofs(resFiles[i]);

        std::string dataLine = "----------------------------------";
        char* buffer = new char[35];
        sprintf_s(buffer, 35, "| id | %-10s | %-4s | %-5s |", "Name", "Year", "Grade");
        std::string dataHeader = std::string(buffer);

        ofs << dataFiles[i] << ":" << std::endl;
        ofs << dataLine << std::endl;
        ofs << dataHeader << std::endl;
        ofs << dataLine << std::endl;
        for (int i = 0; i < studentVector.size(); i++) {
            buffer = new char[50];
            sprintf_s(buffer, 50, "| %2d | %-10s | %4d | %5.2f |",
                i + 1, studentVector.at(i).name,
                studentVector.at(i).year, studentVector.at(i).grade);
            ofs << std::string(buffer) << std::endl;
        }
        ofs << dataLine << std::endl << std::endl;

        ofs << std::endl << std::string(results) << std::endl;
        printf("%s hashes:\n", dataFiles[i].c_str());
        int j = 0;
        while (hashes[j] != '\0') {
            char c = hashes[j];
            printf("%c", hashes[j]);
            j++;
            if (j % 61 == 0)  printf("\n"); 
        }
        printf("\n");

        delete [] students;
        delete[] results;
    }
    return 0;
}

__global__ void hashCounts(Student* data, const int* count, char* results, int* index, char* hashes, int* hashIndex) {
    const auto slice_size = *count / blockDim.x;
    unsigned long start_index = slice_size * threadIdx.x;
    unsigned long end_index;
    if (threadIdx.x == blockDim.x - 1) {
        end_index = *count;
    }
    else {
        end_index = slice_size * (threadIdx.x + 1);
    }

    for (auto i = start_index; i < end_index; i++) {
        char hash[40];
        hashStudent(studentToString(data[i]), hash);
        int k = 0;
        // storing hashes for printing purposes ------------------
        int hto = atomicAdd(hashIndex, 61);
        while (data[i].name[k] != '\0') {
            hashes[hto + k] = data[i].name[k];
            k++;
        }
        for (;k < 20; k++) {
            hashes[hto + k] = ' ';
        }
        hashes[hto + k] = '-';
        k = 0;
        for (;k < 40; k++) {
            hashes[hto + k + 21] = hash[k];
        }
        // --------------------------------------------------------
        int digitsCount = countHashDigits(hash);
        if (digitsCount < 10) {
            int to = atomicAdd(index, 23);
            int j = 0;
            while (data[i].name[j] != '\0') {
                results[to + j] = data[i].name[j];
                j++;
            }
            results[to + j++] = '-';
            results[to + j++] = digitsCount + 48;
            for (;j < 23; j++) {
                results[to + j] = ' ';
            }
        }
    }
}